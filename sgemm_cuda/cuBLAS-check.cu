#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdlib>

#define CHECK_CUDA(call)                                                       \
    {                                                                          \
        hipError_t err = (call);                                              \
        if (err != hipSuccess) {                                              \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__      \
                      << " code=" << err << " \"" << hipGetErrorString(err)  \
                      << "\"" << std::endl;                                    \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

#define CHECK_CUBLAS(call)                                                     \
    {                                                                          \
        hipblasStatus_t stat = (call);                                          \
        if (stat != HIPBLAS_STATUS_SUCCESS) {                                   \
            std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__    \
                      << " code=" << stat << std::endl;                        \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

// Initialize matrix with random float values
void initialize_matrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX * 10.0f;
    }
}

int main() {
    // Use smaller size for testing, adjust up once verified
    int M = 1024, N = 1024, K = 1024;
    float *A = nullptr, *B = nullptr, *C = nullptr;
    float *dA = nullptr, *dB = nullptr, *dC = nullptr;
    float alpha = 1.0f, beta = 0.0f;

    // Allocate host memory
    A = (float *)malloc(M * K * sizeof(float));
    B = (float *)malloc(K * N * sizeof(float));
    C = (float *)malloc(M * N * sizeof(float));
    if (!A || !B || !C) {
        std::cerr << "Host malloc failed\n";
        return EXIT_FAILURE;
    }

    initialize_matrix(A, M, K);
    initialize_matrix(B, K, N);

    // Allocate device memory
    CHECK_CUDA(hipMalloc(&dA, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&dB, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&dC, M * N * sizeof(float)));

    // Copy input matrices to device
    CHECK_CUDA(hipMemcpy(dA, A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dB, B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Record time
    CHECK_CUDA(hipEventRecord(start));

    // Note: Use CUBLAS_OP_T to treat row-major as transposed column-major
    CHECK_CUBLAS(hipblasSgemm(handle,
                             HIPBLAS_OP_T, HIPBLAS_OP_T,
                             M, N, K,
                             &alpha,
                             dB, N,  // B transposed
                             dA, K,  // A transposed
                             &beta,
                             dC, M)); // Result in C

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "cuBLAS SGEMM elapsed time: " << milliseconds << " ms" << std::endl;

    // Copy result back to host
    CHECK_CUDA(hipMemcpy(C, dC, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Cleanup
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipFree(dA));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(dC));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    free(A);
    free(B);
    free(C);

    return 0;
}

