
#include <hip/hip_runtime.h>
#include <stdio.h>


// Write CUDA kernel for naiive matrix multiplication
template <const uint BLOCKSIZE>
__global__ void sgemm_GMEMcoalescing(int M, int N, int K, float alpha, const float *A, 
                            const float *B, float beta, float *C) {

    const uint x = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
    const uint y = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);

    if (x < M && y < N) {
    float tmp = 0.0;
    for (int i = 0; i < K ; ++i){
        tmp += A[ x * K + i ] * B[i*N + y];
    }
    C[x * N + y] = alpha * tmp + beta * C[x*N + y];
    }

}


int main(void) {

  int M=8192, N=8192, K=8192;
  float *A, *B, *C, *dA, *dB, *dC;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  
  A = (float *)malloc(M*K*sizeof(float));
  B = (float *)malloc(K*N*sizeof(float));
  C = (float *)malloc(M*N*sizeof(float));


  hipMalloc(&dA, M*K*sizeof(float));
  hipMalloc(&dB, K*N*sizeof(float));
  hipMalloc(&dC, M*N*sizeof(float));


  hipMemcpy(dA, A, sizeof(float)*M*K, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, sizeof(float)*K*N, hipMemcpyHostToDevice);

  // create as many blocks as necessary to map all of C
  dim3 gridDim((M + 31)/ 32, (N+31)/ 32, 1);
  
  dim3 blockDim(32, 32, 1);
  // launch the asynchronous execution of the kernel on the device
  // Record start time
  hipEventRecord(start);

  // The function call returns immediately on the host
  sgemm_GMEMcoalescing<32><<<gridDim, blockDim>>>(M, N, K, 1.0f, dA, dB, 1.0f, dC);
  
  // Record end time and synchronize
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Copy the result back to Host
  hipMemcpy(C, dC, sizeof(float)*M*N, hipMemcpyDeviceToHost);

  // Calculate elapsed time
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Elapsed time: %f ms\n", milliseconds);

  // Clean up
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  free(A);
  free(B);
  free(C);
}

