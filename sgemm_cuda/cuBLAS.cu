#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// Function to initialize a matrix with random values
void initialize_matrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX * 10.0f; // Random values between 0 and 10
    }
}

int main(void) {

  int M=8192, N=8192, K=8192;
  float *A, *B, *C, *dA, *dB, *dC;
  float alpha= 1.0f, beta= 0.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipblasHandle_t handle;
  hipblasCreate(&handle);


  
  A = (float *)malloc(M*K*sizeof(float));
  B = (float *)malloc(K*N*sizeof(float));
  C = (float *)malloc(M*N*sizeof(float));


  hipMalloc(&dA, M*K*sizeof(float));
  hipMalloc(&dB, K*N*sizeof(float));
  hipMalloc(&dC, M*N*sizeof(float));

  initialize_matrix(A, M, K);
  initialize_matrix(B, K, N);


  hipMemcpy(dA, A, sizeof(float)*M*K, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, sizeof(float)*K*N, hipMemcpyHostToDevice);

  // launch the asynchronous execution of the kernel on the device
  // Record start time
  hipEventRecord(start);

  // The function call returns immediately on the host
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K,
                   &alpha, dA, M, dB, K, &beta, dC, M);
  
  // Record end time and synchronize
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // Copy the result back to Host
  hipMemcpy(C, dC, sizeof(float)*M*N, hipMemcpyDeviceToHost);

  // Calculate elapsed time
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Elapsed time:" << milliseconds << " ms" << std::endl;


  // Clean up
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipblasDestroy(handle);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  free(A);
  free(B);
  free(C);
}

